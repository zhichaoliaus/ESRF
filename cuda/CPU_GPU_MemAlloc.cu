#include <stdio.h>
#include <errno.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <sys/ioctl.h>
#include <string.h>
#include <sys/mman.h>
#include <sys/time.h>
#include <sys/wait.h>
#include <sys/ipc.h>
#include <sys/shm.h>
#include <hip/hip_runtime.h>

__global__ void VecAdd(float* A, float* B, float* C, int N)
{
  for (int i=0; i<N; i++) {
    C[i] = A[i] + B[i];
  }
}

void HostVecAdd(float* A, float* B, float* C, int N)
{
  for (int i=0; i<N; i++) {
    C[i] = A[i] + B[i];
  }
}

void TestHostMemory(int LoopNum)
{
  printf(" - Loop %d:\n", LoopNum+1);
  int N = 1024;
  size_t size = N * sizeof(float);
  float* h_A = (float*)malloc(size);
  float* h_B = (float*)malloc(size);
  float* h_D = (float*)malloc(size);
  // Initialize input vectors
  for (int i = 0; i < N; i++) {
    h_A[i] = (float)((i+1)*rand()%1000)/100;
    h_B[i] = (float)((i+2)*rand()%1000)/100;
  }
  //printf("  => Array A first three data: %f, %f, %f\n", h_A[0], h_A[1], h_A[2]);
  //printf("  => Array B first three data: %f, %f, %f\n", h_B[0], h_B[1], h_B[2]);
  HostVecAdd(h_A, h_B, h_D, N);
  //printf("  => Return D first three data: %f, %f, %f\n", h_D[0], h_D[1], h_D[2]);
}

void TestGPUMemory(int LoopNum)
{
  printf(" - Loop %d:\n", LoopNum+1);
  int N = 1024;
  size_t size = N * sizeof(float);
  float* h_A = (float*)malloc(size);
  float* h_B = (float*)malloc(size);
  float* h_D = (float*)malloc(size);
  // Initialize input vectors
  for (int i = 0; i < N; i++) {
    h_A[i] = (float)((i+1)*rand()%1000)/100;
    h_B[i] = (float)((i+2)*rand()%1000)/100;
  }
  //printf("  => Array A first three data: %f, %f, %f\n", h_A[0], h_A[1], h_A[2]);
  //printf("  => Array B first three data: %f, %f, %f\n", h_B[0], h_B[1], h_B[2]);
  float* d_A, *d_B, *d_D;
  hipMalloc(&d_A, size);
  hipMalloc(&d_B, size);
  hipMalloc(&d_D, size);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  // Invoke kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1)/ threadsPerBlock;
  VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_D, N);
  hipDeviceSynchronize();
  hipMemcpy(h_D, d_D, size, hipMemcpyDeviceToHost);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_D);
  //printf("  => Return D first three data: %f, %f, %f\n", h_D[0], h_D[1], h_D[2]);
  free(h_A);
  free(h_B);
  free(h_D);
}

void TestUnifedMemory(int LoopNum)
{
  printf(" - Loop %d:\n", LoopNum+1);
  int N = 1024;
  size_t size = N * sizeof(float);
  float* h_A, *h_B, *h_D;
  hipMallocManaged((float**)&h_A, size);
  hipMallocManaged((float**)&h_B, size);
  hipMallocManaged((float**)&h_D, size);
  // Initialize input vectors
  for (int i = 0; i < N; i++) {
    h_A[i] = (float)((i+1)*rand()%1000)/100;
    h_B[i] = (float)((i+2)*rand()%1000)/100;
  }
  //printf("  => Array A first three data: %f, %f, %f\n", h_A[0], h_A[1], h_A[2]);
  //printf("  => Array B first three data: %f, %f, %f\n", h_B[0], h_B[1], h_B[2]);
  // Invoke kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1)/ threadsPerBlock;
  VecAdd<<<blocksPerGrid, threadsPerBlock>>>(h_A, h_B, h_D, N);
  hipDeviceSynchronize();
  //printf("  => Return D first three data: %f, %f, %f\n", h_D[0], h_D[1], h_D[2]);
  // Free memory
  hipFree(h_A);
  hipFree(h_B);
  hipFree(h_D);
}

void usage()
{
	printf("Usage: [ options ]\n");
	printf("\t-n <loops>\tRun this number of memcpy loops (default 1)\n");
  printf("\t-m <1|2|3>\tSepecify memory allocation. 1:CPU, 2:GPU, 3: Unified (default 1)\n");
	exit(1);
}

int main(int argc, char *argv[]){
  int LoopCount = 1;
  int MemAllocMode = 1;
  int t;
  struct timeval start, end;
  while (1) {
    int c = getopt(argc, argv, "hn:m:");
    if (c < 0)
      break;
    switch (c) {
    case 'n':
      LoopCount = atoi(optarg);
      break;
    case 'm':
      MemAllocMode = atoi(optarg);
      break;
    case 'h':
      usage();
      break;
    }
  }

  switch (MemAllocMode) {
    case 1:
      printf(">>> Testing Host Memory for %d loop(s)\n", LoopCount);
      gettimeofday(&start, NULL);
      for (int i = 0; i < LoopCount; i++) {
        TestHostMemory(i);
      }
      gettimeofday(&end, NULL);
      break;
    case 2:
      printf(">>> Testing GPU Memory for %d loop(s)\n", LoopCount);
      gettimeofday(&start, NULL);
      for (int i = 0; i < LoopCount; i++) {
        TestGPUMemory(i);
      }
      gettimeofday(&end, NULL);
      break;
    case 3:
      printf(">>> Testing Unified Memory for %d loop(s)\n", LoopCount);
      gettimeofday(&start, NULL);
      for (int i = 0; i < LoopCount; i++) {
        TestUnifedMemory(i);
      }
      gettimeofday(&end, NULL);
      break;
  }
  t = ((end.tv_sec - start.tv_sec)*1000000 + end.tv_usec - start.tv_usec)/LoopCount;
  printf(">>> Average Kernel Runtime is %d uS for %d loops\n", t, LoopCount);
}
